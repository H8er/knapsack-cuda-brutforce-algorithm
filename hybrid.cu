
#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>

#define arraySize 29 //35 max
#define def_div 10
#define W 31
#define threads_per_block 32
#define max_blocks 32

using namespace std;

__constant__ float coefs[arraySize*2];
__global__ void single_thread(float *sh_sum_dev, long int *str_num_dev, float num_of_blocks, int* bdevX,int* global_mem_bin)
{
  float th_w_sum = 0;
   float th_v_sum = 0;
   int th_bin[arraySize];
   int best_bin[arraySize];
  __shared__ float sh_maxs[threads_per_block];
  __shared__ long int indices[threads_per_block];
  int reached = 0;
  indices[threadIdx.x] = blockIdx.x * blockDim.x + threadIdx.x;
  __syncthreads();

long signed int num_to_bin = blockIdx.x * blockDim.x + threadIdx.x;
//num_to_bin += max_blocks * n_of_it;
#pragma unroll
  for (uint i = 0; i < def_div; i++)
    {
      th_bin[i] = ((num_to_bin) >> i) % 2;
      th_w_sum += th_bin[i] * coefs[i];
      th_v_sum += th_bin[i] * coefs[i+arraySize];
      best_bin[i] = th_bin[i];
    }
#pragma unroll
    for (uint i = def_div; i < arraySize; i++)
      {
        th_bin[i] = -1;
      }
__syncthreads ();
register int Capacity = W - th_w_sum;
sh_maxs[threadIdx.x] = (th_w_sum > W) ? 0:th_v_sum;
__syncthreads ();


//H_S
int h = def_div;
long int ns = 0;
bool forward;

while(h-def_div!=-1){
  ns++;
  forward = true;
  if(th_bin[h]==-1){
     th_bin[h]=1;
  }else{
  if(th_bin[h]==1){
     th_bin[h]=0;
  }else{
  if(th_bin[h]==0){
     th_bin[h]=-1;
    h--;
    forward=false;
  }
}
}
  if(h==arraySize-1){
    int cw = 0;
    int cp = 0;
    #pragma unroll
    for(int i = def_div;i<arraySize;i++){
      cp += coefs[i+arraySize] * th_bin[i];
      cw += coefs[i] * th_bin[i];
    }
    if((cw <= Capacity) &&(cp > reached)){
      reached = cp;
      #pragma unroll
      for(int i = 0; i < arraySize; i++){
        best_bin[i] = th_bin[i];
      }
    }
  }
  else{
    int cw = 0;
    for(int i = def_div ; i < arraySize; i++){
      cw += coefs[i] * th_bin[i];
    }
    if (cw > Capacity) forward = false;
    cw = 0;
    float cp = 0;
    int nw = 0;
    int np = 0;
    #pragma unroll
    for(int i = def_div;i < arraySize;i++){
      np = th_bin[i]!=-1? th_bin[i] * coefs[i+arraySize]:coefs[i+arraySize];
      nw = th_bin[i]!=-1? th_bin[i] * coefs[i]: coefs[i];
      if(cw+nw <= Capacity){
        cw += nw;
        cp += np;
      }
      else{
        cp+=np*(Capacity-cw)/nw;
        break;
      }
    }
    int b = cp;
    if (b <= reached){
      forward = false;
    }
  }
  if(forward){if(h<arraySize-1){h++;}
              }
  }

sh_maxs[threadIdx.x] += reached;




__syncthreads();
//reduction on block
  for (uint offset = blockDim.x >> 1; offset >= 1; offset >>= 1)
    {
      if (threadIdx.x < offset)
	{
	  if (sh_maxs[threadIdx.x] < sh_maxs[threadIdx.x + offset])
	    {
	      sh_maxs[threadIdx.x] = sh_maxs[threadIdx.x + offset];
	      indices[threadIdx.x] = indices[threadIdx.x + offset];
	    }
	}
      __syncthreads ();
    }
  // write result for this block to global mem
  if(threadIdx.x == 0){
  sh_sum_dev[blockIdx.x] = sh_maxs[0];
  str_num_dev[blockIdx.x] = indices[0];
  }
  if(blockIdx.x*blockDim.x+threadIdx.x == indices[0]){
    #pragma unroll
    for(int i = 0; i< arraySize;i++){
      global_mem_bin[blockIdx.x*arraySize + i] = best_bin[i];
  }
  }
  __syncthreads();

}

__global__ void
reduction_max (float *s, long int *str_num_dev,int* global_mem_bin)
{
  int ID = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ int sdata[threads_per_block*2];
  sdata[threadIdx.x] = s[ID];
  sdata[threadIdx.x + threads_per_block] = str_num_dev[ID];

  __syncthreads ();
  // do reduction in shared mem
  for (uint s = blockDim.x >>1; s > 0; s >>= 1)
    {
      if (threadIdx.x < s)
	{
	  if (sdata[threadIdx.x] < sdata[threadIdx.x + s])
	    {
	      sdata[threadIdx.x] = sdata[threadIdx.x + s];
	      sdata[threadIdx.x + threads_per_block] =
		sdata[threadIdx.x + threads_per_block + s];
	    }
	}
      __syncthreads ();
    }
  // write result for this block to global mem
  if (threadIdx.x == 0)
    {
			//if(sdata[0]>s[0]){//}&&(blockIdx.x>0)){
      s[blockIdx.x] = sdata[0];
      str_num_dev[blockIdx.x] = sdata[threads_per_block];

            #pragma unroll
            for(int i = 0; i < arraySize;i++){
              global_mem_bin[i] = global_mem_bin[(sdata[threads_per_block]-arraySize)/threads_per_block*arraySize + i];
          }
		}

}


__global__ void
which_string (long int a, int *view_dev,int* global_mem_bin)
{
  view_dev[threadIdx.x] = global_mem_bin[blockIdx.x*arraySize + threadIdx.x];
}


void quickSortR(float* a,float* b, long N) {
// На входе - массив a[], a[N] - его последний элемент.

    long i = 0, j = N;      // поставить указатели на исходные места
    float temp, p;

    p = a[ N>>1 ];      // центральный элемент

    // процедура разделения
    do {
        while ( a[i] > p ) i++;
        while ( a[j] < p ) j--;

        if (i <= j) {
            temp = a[i]; a[i] = a[j]; a[j] = temp;
            temp = b[i]; b[i] = b[j]; b[j] = temp;
            temp = b[i+arraySize]; b[i+arraySize] = b[j+arraySize]; b[j+arraySize] = temp;
            i++; j--;
        }
    } while ( i<=j );

    // рекурсивные вызовы, если есть, что сортировать
    if ( j > 0 ) quickSortR(a,b, j);
    if ( N > i ) quickSortR(a+i,b+i, N-i);
}



    int main(){

      long int strSize_b = pow (2, arraySize);
      int num_of_blocks = strSize_b / threads_per_block;
      float *Sum = new float[32];	// = { 0 };
      float *sh_sum_dev;
      //float weight[31] ={ 5, 10, 17, 19, 20, 23, 26, 30, 32, 38, 40, 44, 47, 50, 55, 56, 56, 60, 62, 66, 70, 75, 77, 80, 81, 90,93,96,101,107,115 };
      //float values[31] ={ 10, 13, 16, 22, 30, 25, 55, 90, 110, 115, 130, 120, 150, 170, 194, 199, 194, 199, 217, 230, 248, 250, 264, 271, 279, 286,293,299,305,313,321 };
      float dev_coefs[62] = {5, 10, 17, 19, 20, 23, 26, 30, 32, 38, 40, 44, 47, 50, 55, 56, 56, 60, 62, 66, 70, 75, 77, 80, 81, 90,93,96,101,107,115, 10, 13, 16, 22, 30, 25, 55, 90, 110, 115, 130, 120, 150, 170, 194, 199, 194, 199, 217, 230, 248, 250, 264, 271, 279, 286,293,299,305,313,321 };
      //float dev_coefs[60] = {5, 10, 17, 19, 20, 23, 26, 30, 32, 38, 40, 44, 47, 50, 55, 56, 56, 60, 62, 66, 70, 75, 77, 80, 81, 90,93,96,101,107, 10, 13, 16, 22, 30, 25, 55, 90, 110, 115, 130, 120, 150, 170, 194, 199, 194, 199, 217, 230, 248, 250, 264, 271, 279, 286,293,299,305,313 };
      //float dev_coefs[58] = {5, 10, 17, 19, 20, 23, 26, 30, 32, 38, 40, 44, 47, 50, 55, 56, 56, 60, 62, 66, 70, 75, 77, 80, 81, 90,93,96,101, 10, 13, 16, 22, 30, 25, 55, 90, 110, 115, 130, 120, 150, 170, 194, 199, 194, 199, 217, 230, 248, 250, 264, 271, 279, 286,293,299,305 };

      //float *values_dev;
      long int *str_num_dev;
      long int *str_num = new long int[1];
      float N_of_rep;
      N_of_rep = num_of_blocks/max_blocks;
      cout <<"N of items "<<arraySize<<"\n";
      cout<<"N of blocks "<<num_of_blocks<<"\n";
      cout<<"strSize_b = "<<strSize_b<<"\n";
      cout<<"num_of_blocks / threads_per_block = "<<num_of_blocks / threads_per_block<<"\n";
      cout<<"N of repeats = "<<N_of_rep<<"\n";
      cout<<"sing param = "<<num_of_blocks/N_of_rep<<" _ "<< threads_per_block<<"\n";
      cout<<"red param "<<num_of_blocks / threads_per_block<<"  ,  "<<strSize_b/num_of_blocks<<"\n";

      float* additional_array = new float[arraySize];
      for(int i = 0; i < arraySize;i++){
      additional_array[i] = dev_coefs[i+arraySize]/dev_coefs[i];
      }

      quickSortR(additional_array,dev_coefs,arraySize-1);

      for(int i = 0;i<arraySize*2;i++){dev_coefs[i] = 2;}

      std::chrono::time_point<std::chrono:: high_resolution_clock> start, end;
          start = std::chrono::high_resolution_clock::now();

      int* bdevX;
      hipMalloc ((void **) &bdevX, arraySize * sizeof (int));
      int* global_mem_bin;
      hipMalloc ((void **) &global_mem_bin, max_blocks*arraySize * sizeof (int));





      hipMalloc ((void **) &sh_sum_dev,  num_of_blocks * sizeof (float));
      hipMalloc ((void **) &str_num_dev, num_of_blocks * sizeof (float));
      hipMemcpyToSymbol (HIP_SYMBOL(coefs), dev_coefs, 2*arraySize * sizeof (float));

    //int sing_blocks = num_of_blocks/N_of_rep;

            //for(int i = 0;i<N_of_rep;i++){
              //cout<<i;
      single_thread <<< max_blocks, threads_per_block >>> (sh_sum_dev, str_num_dev, num_of_blocks,bdevX,global_mem_bin);
                 //}



reduction_max<<<1,max_blocks>>>(sh_sum_dev,str_num_dev,global_mem_bin);
int* suda = new int[arraySize];
      hipMemcpy (Sum, sh_sum_dev, sizeof (int), hipMemcpyDeviceToHost);
      hipMemcpy (str_num, str_num_dev, sizeof (long int), hipMemcpyDeviceToHost);
      hipMemcpy (suda, global_mem_bin, arraySize*sizeof (int), hipMemcpyDeviceToHost);


      end = std::chrono:: high_resolution_clock::now();

          int elapsed_seconds = std::chrono::duration_cast<std::chrono::microseconds>
                                   (end-start).count();
          std::time_t end_time = std::chrono::system_clock::to_time_t(end);

          std::cout<< "Время выполнения: " << elapsed_seconds << "microseconds\n";

      cout << "Acheived maximal sum = " << Sum[0] << "\n";
      cout<<str_num[0]<<"\n";
        int *view = new int[arraySize];
        //int *view_dev;
        //cudaMalloc ((void **) &view_dev, arraySize * sizeof (int));
        //which_string <<< 1, arraySize >>> (str_num[0], view_dev,global_mem_bin);
        //cudaMemcpy (view, view_dev, arraySize * sizeof (int),
      	  //    cudaMemcpyDeviceToHost);
        for (int i = 0; i < arraySize; i++)
          {
            cout << suda[i] << " ";
          } cout << "\n";
        //check
        int checksum = 0;
        for (int i = 0; i < arraySize; i++)
          {
            checksum += dev_coefs[i+arraySize] * suda[i];
          }
        cout << "Validation sum = " << checksum << "\n";
        checksum = 0;
        for (int i = 0; i < arraySize; i++)
          {
            checksum += dev_coefs[i] * suda[i];
          } cout << "Weight = " << checksum << "\n";



        hipFree(coefs);
        hipFree (sh_sum_dev);
        hipFree (str_num_dev);
        hipFree(bdevX);
        hipFree(global_mem_bin);

        delete [] Sum;
        delete [] str_num;
        delete [] additional_array;

        cout<<"Проверка. CPU version:\n";
        int *X = new int[arraySize];
        int *bestX = new int[arraySize];
        for(int i = 0; i < arraySize; i++){
          X[i] = -1;
          bestX[i] = 0;
        }
        int curr_sum = 0;
        int reached_max = 0;

        float *cpu_bin = new float[arraySize];

        start = std::chrono::high_resolution_clock::now();

        int h = 0;
        int k = h;//def_div;
        long int ns = 0;
        bool forward;
        while(h-k!=-1){
          ns++;
          forward = true;
          if(X[h]==-1){
            X[h]=1;
          }else{
          if(X[h]==1){
            X[h]=0;
          }else{
          if(X[h]==0){
            X[h]=-1;
            h--;
            forward=false;
          }
        }
        }
          if(h==arraySize-1){
            int cw = 0;
            int cp = 0;
            for(int i = k;i<arraySize;i++){
              cp += dev_coefs[i+arraySize]*X[i];
              cw += dev_coefs[i]*X[i];
            }
            if((cw <= W) &&(cp > reached_max)){
              reached_max = cp;
              for(int i = k; i < arraySize; i++){
                bestX[i] = X[i];
              }
            }
          }
          else{
            int cw = 0;
            for(int i = k ; i < arraySize; i++){
              cw += dev_coefs[i]*X[i];
            }
            if (cw > W) forward = false;
            cw = 0;
            float cp = 0;
            int nw = 0;
            int np = 0;
            for(int i = k;i<arraySize;i++){
              np = X[i]!=-1? X[i] * dev_coefs[i+arraySize]:dev_coefs[i+arraySize];
              nw = X[i]!=-1? X[i] * dev_coefs[i]: dev_coefs[i];
              if(cw+nw <= W){
                cw += nw;
                cp += np;
              }
              else{
                cp+=np*(W-cw)/nw;
                break;
              }
            }
            int b = cp;
            if (b <= reached_max){
              forward = false;
            }
          }
          if(forward){if(h<arraySize-1){h++;}}
          }


          end = std::chrono:: high_resolution_clock::now();

              elapsed_seconds = std::chrono::duration_cast<std::chrono::microseconds>
                                       (end-start).count();
               end_time = std::chrono::system_clock::to_time_t(end);

          std::cout<< "Время выполнения: " << elapsed_seconds << "microseconds\n";

        cout<<"MAX = "<<reached_max<<"\n";
        for(int k = 0 ; k < arraySize;k++){
        cout<<bestX[k];
        curr_sum += bestX[k]*dev_coefs[k+arraySize];
        }cout<<"\nЧисло итераций = "<<ns<<"\n";

delete [] suda;

return 0;
}

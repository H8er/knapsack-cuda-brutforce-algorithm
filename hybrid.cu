
#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>

#define arraySize 31 //35 max
#define def_div 5
#define W 500
#define threads_per_block 32
#define max_blocks 32768

using namespace std;

__constant__ float coefs[arraySize*2];
__global__ void single_thread(float *sh_sum_dev, float *str_num_dev, float num_of_blocks, int* bdevX)
{
  float th_w_sum = 0;
   float th_v_sum = 0;
   int th_bin[arraySize];
   int best_bin[arraySize];
  __shared__ float sh_maxs[threads_per_block];
  __shared__ int indices[threads_per_block];
  int reached = 0;
  indices[threadIdx.x] = blockIdx.x * blockDim.x + threadIdx.x;
  __syncthreads();

long signed int num_to_bin = blockIdx.x * blockDim.x + threadIdx.x;
//num_to_bin += max_blocks * n_of_it;
#pragma unroll
  for (uint i = 0; i < def_div; i++)
    {
      th_bin[i] = ((num_to_bin) >> i) % 2;
      th_w_sum += th_bin[i] * coefs[i];
      th_v_sum += th_bin[i] * coefs[i+arraySize];
      best_bin[i] = th_bin[i];
    }
#pragma unroll
    for (uint i = def_div; i < arraySize; i++)
      {
        th_bin[i] = -1;
      }
__syncthreads ();
register int Capacity = W - th_w_sum;
sh_maxs[threadIdx.x] = (th_w_sum > W) ? 0:th_v_sum;
__syncthreads ();


//H_S
int h = def_div;
long int ns = 0;
bool forward;

while(h-def_div!=-1){
  ns++;
  forward = true;
  if(th_bin[h]==-1){
     th_bin[h]=1;
  }else{
  if(th_bin[h]==1){
     th_bin[h]=0;
  }else{
  if(th_bin[h]==0){
     th_bin[h]=-1;
    h--;
    forward=false;
  }
}
}
  if(h==arraySize-1){
    int cw = 0;
    int cp = 0;
    //#pragma unroll
    for(int i = def_div;i<arraySize;i++){
      cp += coefs[i+arraySize] * th_bin[i];
      cw += coefs[i] * th_bin[i];
    }
    if((cw <= Capacity) &&(cp > reached)){
      reached = cp;
      #pragma unroll
      for(int i = 0; i < arraySize; i++){
        best_bin[i] = th_bin[i];
      }
    }
  }
  else{
    int cw = 0;
    for(int i = def_div ; i < arraySize; i++){
      cw += coefs[i] * th_bin[i];
    }
    if (cw > Capacity) forward = false;
    cw = 0;
    float cp = 0;
    int nw = 0;
    int np = 0;
    #pragma unroll
    for(int i = def_div;i < arraySize;i++){
      np = th_bin[i]!=-1? th_bin[i] * coefs[i+arraySize]:coefs[i+arraySize];
      nw = th_bin[i]!=-1? th_bin[i] * coefs[i]: coefs[i];
      if(cw+nw <= Capacity){
        cw += nw;
        cp += np;
      }
      else{
        cp+=np*(Capacity-cw)/nw;
        break;
      }
    }
    int b = cp;
    if (b <= reached){
      forward = false;
    }
  }
  if(forward){if(h<arraySize-1){h++;}
              }
  }

sh_maxs[threadIdx.x] += reached;
__syncthreads();
//reduction on block
  for (uint offset = blockDim.x >> 1; offset >= 1; offset >>= 1)
    {
      if (threadIdx.x < offset)
	{
	  if (sh_maxs[threadIdx.x] < sh_maxs[threadIdx.x + offset])
	    {
	      sh_maxs[threadIdx.x] = sh_maxs[threadIdx.x + offset];
	      indices[threadIdx.x] = indices[threadIdx.x + offset];
	    }
	}
      __syncthreads ();
    }
  // write result for this block to global mem
  if(threadIdx.x == 0){
  sh_sum_dev[blockIdx.x] = sh_maxs[0];
  //str_num_dev[blockIdx.x] = indices[0];
  }
  //__syncthreads();
  if(threadIdx.x == indices[0]){
    #pragma unroll
  for(int i = 0;i<arraySize;i++){bdevX[i] = best_bin[i];}
  }
/*
  if(threadIdx.x<arraySize){
  bdevX[threadIdx.x] = Capacity;
}
*/
}

__global__ void
reduction_max (float *s, float *str_num_dev)
{
  int ID = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ float sdata[threads_per_block*2];
  sdata[threadIdx.x] = s[ID];
  sdata[threadIdx.x + threads_per_block] = str_num_dev[ID];

  __syncthreads ();
  // do reduction in shared mem
  for (uint s = blockDim.x >>1; s > 0; s >>= 1)
    {
      if (threadIdx.x < s)
	{
	  if (sdata[threadIdx.x] < sdata[threadIdx.x + s])
	    {
	      sdata[threadIdx.x] = sdata[threadIdx.x + s];
	      sdata[threadIdx.x + threads_per_block] =
		sdata[threadIdx.x + threads_per_block + s];
	    }
	}
      __syncthreads ();
    }
  // write result for this block to global mem
  if (threadIdx.x == 0)
    {
			//if(sdata[0]>s[0]){//}&&(blockIdx.x>0)){
      s[blockIdx.x] = sdata[0];
      str_num_dev[blockIdx.x] = sdata[threads_per_block];
		}
    //}
}


__global__ void
which_string (int a, int *view_dev)
{
  view_dev[threadIdx.x] = (a >> threadIdx.x) % 2;
}
///H-S
void quickSortR(float* a,float* b, long N) {
// На входе - массив a[], a[N] - его последний элемент.

    long i = 0, j = N;      // поставить указатели на исходные места
    float temp, p;

    p = a[ N>>1 ];      // центральный элемент

    // процедура разделения
    do {
        while ( a[i] > p ) i++;
        while ( a[j] < p ) j--;

        if (i <= j) {
            temp = a[i]; a[i] = a[j]; a[j] = temp;
            temp = b[i]; b[i] = b[j]; b[j] = temp;
            temp = b[i+arraySize]; b[i+arraySize] = b[j+arraySize]; b[j+arraySize] = temp;
            i++; j--;
        }
    } while ( i<=j );

    // рекурсивные вызовы, если есть, что сортировать
    if ( j > 0 ) quickSortR(a,b, j);
    if ( N > i ) quickSortR(a+i,b+i, N-i);
}


    int main(){

      long int strSize_b = pow (2, arraySize);
      int num_of_blocks = strSize_b / threads_per_block;
      float *Sum = new float[32];	// = { 0 };
      float *sh_sum_dev;
      //float weight[31] ={ 5, 10, 17, 19, 20, 23, 26, 30, 32, 38, 40, 44, 47, 50, 55, 56, 56, 60, 62, 66, 70, 75, 77, 80, 81, 90,93,96,101,107,115 };
      //float values[31] ={ 10, 13, 16, 22, 30, 25, 55, 90, 110, 115, 130, 120, 150, 170, 194, 199, 194, 199, 217, 230, 248, 250, 264, 271, 279, 286,293,299,305,313,321 };
      float dev_coefs[62] = {5, 10, 17, 19, 20, 23, 26, 30, 32, 38, 40, 44, 47, 50, 55, 56, 56, 60, 62, 66, 70, 75, 77, 80, 81, 90,93,96,101,107,115, 10, 13, 16, 22, 30, 25, 55, 90, 110, 115, 130, 120, 150, 170, 194, 199, 194, 199, 217, 230, 248, 250, 264, 271, 279, 286,293,299,305,313,321 };
      //float dev_coefs[60] = {5, 10, 17, 19, 20, 23, 26, 30, 32, 38, 40, 44, 47, 50, 55, 56, 56, 60, 62, 66, 70, 75, 77, 80, 81, 90,93,96,101,107, 10, 13, 16, 22, 30, 25, 55, 90, 110, 115, 130, 120, 150, 170, 194, 199, 194, 199, 217, 230, 248, 250, 264, 271, 279, 286,293,299,305,313 };
      //float dev_coefs[58] = {5, 10, 17, 19, 20, 23, 26, 30, 32, 38, 40, 44, 47, 50, 55, 56, 56, 60, 62, 66, 70, 75, 77, 80, 81, 90,93,96,101, 10, 13, 16, 22, 30, 25, 55, 90, 110, 115, 130, 120, 150, 170, 194, 199, 194, 199, 217, 230, 248, 250, 264, 271, 279, 286,293,299,305 };

      //float *values_dev;
      float *str_num_dev;
      float *str_num = new float[1];
      float N_of_rep;
      N_of_rep = num_of_blocks/max_blocks;
      cout <<"N of items "<<arraySize<<"\n";
      cout<<"N of blocks "<<num_of_blocks<<"\n";
      cout<<"strSize_b = "<<strSize_b<<"\n";
      cout<<"num_of_blocks / threads_per_block = "<<num_of_blocks / threads_per_block<<"\n";
      cout<<"N of repeats = "<<N_of_rep<<"\n";
      cout<<"sing param = "<<num_of_blocks/N_of_rep<<" _ "<< threads_per_block<<"\n";
      cout<<"red param "<<num_of_blocks / threads_per_block<<"  ,  "<<strSize_b/num_of_blocks<<"\n";

      float* additional_array = new float[arraySize];
      for(int i = 0; i < arraySize;i++){
      additional_array[i] = dev_coefs[i+arraySize]/dev_coefs[i];
      }

      quickSortR(additional_array,dev_coefs,arraySize-1);
      cout<<"\n   ";
      for(int i = 0;i<arraySize;i++){
        cout<<dev_coefs[i+arraySize]<<" ";
      }cout<<"\n    ";
      for(int i = 0;i<arraySize;i++){
        cout<<dev_coefs[i]<<" ";
      }cout<<"\n";


      std::chrono::time_point<std::chrono:: high_resolution_clock> start, end;
          start = std::chrono::high_resolution_clock::now();

      int* bdevX;
      hipMalloc ((void **) &bdevX, arraySize * sizeof (int));


    //for(int i = 0;i<arraySize*2;i++){dev_coefs[i] = 2;}



      hipMalloc ((void **) &sh_sum_dev,  num_of_blocks * sizeof (float));
      hipMalloc ((void **) &str_num_dev, num_of_blocks * sizeof (float));
      hipMemcpyToSymbol (HIP_SYMBOL(coefs), dev_coefs, 2*arraySize * sizeof (float));

    //int sing_blocks = num_of_blocks/N_of_rep;

            //for(int i = 0;i<N_of_rep;i++){
              //cout<<i;
      single_thread <<< 32, 32 >>> (sh_sum_dev, str_num_dev, num_of_blocks,bdevX);
                 //}
                 hipMemcpy (Sum, sh_sum_dev, 32*sizeof (int), hipMemcpyDeviceToHost);
                 for(int i = 0; i < 32;i++){cout<<Sum[i]<<" ";}cout<<"\n";
reduction_max<<<1,32>>>(sh_sum_dev,str_num_dev);
int* suda = new int[arraySize];
      hipMemcpy (Sum, sh_sum_dev, 32*sizeof (int), hipMemcpyDeviceToHost);
      hipMemcpy (suda, bdevX, arraySize * sizeof (int), hipMemcpyDeviceToHost);



      end = std::chrono:: high_resolution_clock::now();

          int elapsed_seconds = std::chrono::duration_cast<std::chrono::microseconds>
                                   (end-start).count();
          std::time_t end_time = std::chrono::system_clock::to_time_t(end);

          std::cout<< "Время выполнения: " << elapsed_seconds << "microseconds\n";

      cout << "Acheived maximal sum = " << Sum[0] << "\n";
      for (int i = 0; i < arraySize; i++)
        {
          cout << suda[i] << " ";
        } cout << "\n";

        hipFree(coefs);
        hipFree (sh_sum_dev);
        hipFree (str_num_dev);
        hipFree(bdevX);

        delete [] Sum;
        delete [] str_num;
        delete [] additional_array;

        cout<<"Проверка. CPU version:\n";
        int *X = new int[arraySize];
        int *bestX = new int[arraySize];
        for(int i = 0; i < arraySize; i++){
          X[i] = -1;
          bestX[i] = 0;
        }
        int curr_sum = 0;
        int reached_max = 0;

        float *cpu_bin = new float[arraySize];

        start = std::chrono::high_resolution_clock::now();

        int h = 0;
        int k = h;//def_div;
        long int ns = 0;
        bool forward;
        while(h-k!=-1){
          ns++;
          forward = true;
          if(X[h]==-1){
            X[h]=1;
          }else{
          if(X[h]==1){
            X[h]=0;
          }else{
          if(X[h]==0){
            X[h]=-1;
            h--;
            forward=false;
          }
        }
        }
          if(h==arraySize-1){
            int cw = 0;
            int cp = 0;
            for(int i = k;i<arraySize;i++){
              cp += dev_coefs[i+arraySize]*X[i];
              cw += dev_coefs[i]*X[i];
            }
            if((cw <= W) &&(cp > reached_max)){
              reached_max = cp;
              for(int i = k; i < arraySize; i++){
                bestX[i] = X[i];
              }
            }
          }
          else{
            int cw = 0;
            for(int i = k ; i < arraySize; i++){
              cw += dev_coefs[i]*X[i];
            }
            if (cw > W) forward = false;
            cw = 0;
            float cp = 0;
            int nw = 0;
            int np = 0;
            for(int i = k;i<arraySize;i++){
              np = X[i]!=-1? X[i] * dev_coefs[i+arraySize]:dev_coefs[i+arraySize];
              nw = X[i]!=-1? X[i] * dev_coefs[i]: dev_coefs[i];
              if(cw+nw <= W){
                cw += nw;
                cp += np;
              }
              else{
                cp+=np*(W-cw)/nw;
                break;
              }
            }
            int b = cp;
            if (b <= reached_max){
              forward = false;
            }
          }
          if(forward){if(h<arraySize-1){h++;}}
          }


          end = std::chrono:: high_resolution_clock::now();

              elapsed_seconds = std::chrono::duration_cast<std::chrono::microseconds>
                                       (end-start).count();
               end_time = std::chrono::system_clock::to_time_t(end);

          std::cout<< "Время выполнения: " << elapsed_seconds << "microseconds\n";

        cout<<"MAX = "<<reached_max<<"\n";
        for(int k = 0 ; k < arraySize;k++){
        cout<<bestX[k];
        curr_sum += bestX[k]*dev_coefs[k+arraySize];
        }cout<<"\nЧисло итераций = "<<ns<<"\n";

delete [] suda;

return 0;
}

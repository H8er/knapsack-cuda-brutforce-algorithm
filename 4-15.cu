#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <cstdlib>
#include <stdlib.h>
#include <sys/time.h>
#include <iostream>


using namespace std;

__global__ void T_binary(int*bin_dev, int *_del) {
	int bli = blockIdx.x * blockDim.x;
	int idx = threadIdx.x;
	bin_dev[bli + idx] = blockIdx.x / _del[idx] % 2;
}

__global__ void bin_multiplication(int *bin_dev, int* wight_dev, int *s_dev, int*values_dev)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	s_dev[i] = bin_dev[i] * values_dev[threadIdx.x];//s_dev -> prices*bin
	bin_dev[i] = bin_dev[i] * wight_dev[threadIdx.x];//bin_dev -> weights*bin
}

__global__ void summing(int* in_dev, int* sums) {
	extern __shared__ int sdata[];
	// each thread loads one element from global to shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	sdata[tid] = in_dev[i];
	//__syncthreads();
	//unroled summing is faster than reduction according to my tests
#pragma unroll
for(ushort  i = 1;i < 16;i++){
	sdata[0]+=sdata[i];
}
		__syncthreads();
	// write result for this block to global mem
	sums[blockIdx.x] = sdata[0];
}

__global__ void additional_summing(int *whatToAdd,int *whereToAdd){
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	whereToAdd[i+32768] = whereToAdd[i]+whatToAdd[15];
}


__global__ void zeroing(int *w, int *s, int W) {
	int bli = blockIdx.x * blockDim.x;
	int idx = threadIdx.x;
	if (w[bli+idx] > W) { s[bli + idx] = 0; w[bli + idx] = 0; }
}
__global__ void reduction_max(int* s) {
	extern __shared__ int sdata[];
	// each thread loads one element from global to shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	sdata[tid] = s[i];
	__syncthreads();
	// do reduction in shared mem
	for (unsigned int s = blockDim.x / 2; s>0; s >>= 1) {
		if (tid < s) {
			if (sdata[tid] < sdata[tid + s])
				sdata[tid] = sdata[tid + s];
		}
		__syncthreads();
	}
	// write result for this block to global mem
	if (tid == 0) s[blockIdx.x] = sdata[0];
}

__global__ void kermax2(int *s, int N) {
	//__shared__ int max[32];
	unsigned int tid = threadIdx.x;
	int off = N / 2;
	if (tid < off) {
		if (s[tid] < s[tid + off]) {
			s[tid] = s[tid + off];
		}
	}
}

int main()
{
		int W = 350;
		int arraySize;
		cout<<"Enter size of array (6-15): ";
		cin>>arraySize;		

		struct timeval t0,t1;
			gettimeofday(&t0, NULL);

		int totalSize = arraySize*pow(2,arraySize);
		int strSize_b = pow(2,arraySize);
		int flag=0;
		if (arraySize>15){
			strSize_b/=(pow(2,(arraySize-15)));
			flag=1;
		}
		int *Sum=new int[totalSize];// = { 0 };
		int *s;
		int *bin_dev;
		int *wight_dev;
		int wight[16] = { 5,10,17,19,20, 23,26,30,32,38, 40,44,47,50,55,56 };// 55, 56, 60, 62, 66, 70	};
		int values[16] = { 10,13,16,22,30, 25,55,90,110,115, 130,120,150,170,194,199 };// , 194, 199, 217, 230, 248	};
		int *w;
		int *values_dev;

		int *del = new int[arraySize], *dev_del;
		hipMalloc((void**)&dev_del, arraySize * sizeof(int));
		for (int i = 0; i < arraySize; i++) {
			del[i] = pow(2, i);
		}

		hipMemcpy(dev_del, del, arraySize * sizeof(int), hipMemcpyHostToDevice);
		hipMalloc((void**)&bin_dev, totalSize * sizeof(int));

		int*s_dev;
		hipMalloc((void**)&s_dev, totalSize * sizeof(int));
		hipMalloc((void**)&wight_dev, arraySize * sizeof(int));
		hipMalloc((void**)&s, totalSize * sizeof(int));
		hipMalloc((void**)&values_dev, arraySize * sizeof(int));
		hipMalloc((void**)&w, totalSize * sizeof(int));
		hipMemcpy(wight_dev, wight, arraySize * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(values_dev, values, arraySize * sizeof(int), hipMemcpyHostToDevice);

//creating of binary table
		T_binary << <strSize_b, arraySize >> > (bin_dev, dev_del);

//multiplication of weight and value parameters of each item on binary table strings
		bin_multiplication << <strSize_b, arraySize >> > (bin_dev, wight_dev, s_dev, values_dev);

		summing << <strSize_b, arraySize,arraySize*sizeof(int) >> > (bin_dev, w);
		summing << <strSize_b, arraySize,arraySize*sizeof(int) >> > (s_dev, s);

		int a=strSize_b/1024;
		int b = 1024;
		if (a==0){
			a=1;
			b = pow(2,arraySize);}

		//additional actions if flag==1
		if(flag==1){
additional_summing<<<a, b>>>(wight_dev,w);
additional_summing<<<a, b>>>(values_dev,s);
		}

//zeroing of unsuitable item's combinations



		zeroing << <a, b >> > (w, s, W);

//finding maximal value for each block
		reduction_max << <a,b,b*sizeof(int) >> > (s);
		hipMemcpy(Sum, s, a*sizeof(int), hipMemcpyDeviceToHost);
		for(int i=0;i<a;i++){cout<<Sum[i]<<" ";}
//second step of finding maximal value
		for (int i = 32; i >= 1; i /= 2) {
			kermax2 << <1, i >> > (s,i);
		}

		hipMemcpy(Sum, s, sizeof(int), hipMemcpyDeviceToHost);

		cout <<"\n"<<"GPU max = " << Sum[0];

		hipFree(bin_dev);
		hipFree(wight_dev);
		hipFree(s);
		hipFree(w);
		hipFree(s_dev);

		//CPU version
		float fTimeStart = clock() / (float)(CLOCKS_PER_SEC);
		int **bin = new int*[strSize_b];
			for(int i=0;i<strSize_b;i++){
				bin[i] = new int[arraySize];
			}
		int k = 0;
		for (int i = 0; i < strSize_b; i++) {
			k = i;
			for (int j = 0; j <arraySize; j++) {
				bin[i][j] = k % 2;
				k /= 2;
			}
		}

		int **prices  = new int*[strSize_b];
		int **weights = new int*[strSize_b];
			for(int i=0;i<strSize_b;i++){
				prices[i]  = new int[arraySize];
				weights[i] = new int[arraySize];
			}

		int *Sweig = new int[strSize_b];
		int *Sval = new int[strSize_b];
		for (int i = 0; i < strSize_b; i++) {
			for (int j = 0; j < arraySize; j++) {
				weights[i][j] = wight[j] * bin[i][j];
				prices[i][j] = values[j] * bin[i][j];
			}
		}
		for (int i = 0; i < strSize_b; i++) {
			for (int j = 0; j < arraySize; j++) {
				Sweig[i] += weights[i][j];
				Sval[i] += prices[i][j];
			}
		}
		int max = 0; k = 0;
		for (int i = 0; i < strSize_b; i++) {
			if ((Sweig[i] <= W) && (Sval[i] > max)) {
				k = i; max = Sval[i];
			}
		}
		float fTimeStop = clock() / (float)CLOCKS_PER_SEC;
		cout << "   CPU max = " << max << "\n";
		//cout << "CPU time is " << (fTimeStop - fTimeStart) * 1000 << " milli-seconds\n";
//memory freeing
		for(int i=0;i<strSize_b;i++){
			delete [] bin[strSize_b];
			delete [] prices[strSize_b];
			delete [] weights[strSize_b];
		}
		delete [] Sweig;
		delete [] Sval;

gettimeofday(&t1, 0);
long sec = (t1.tv_sec-t0.tv_sec);
long usec =  t1.tv_usec-t0.tv_usec;
cout<<sec<<","<<usec;
return 0;
}

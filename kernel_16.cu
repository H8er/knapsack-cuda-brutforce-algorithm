#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <cstdlib>
#include <stdlib.h>
#include <ctime>
#include <time.h>
#include <iostream>


using namespace std;

__global__ void T_binary3(int*bin_dev, int *_del) {
	int bli = blockIdx.x * blockDim.x;
	int idx = threadIdx.x;
	bin_dev[bli + idx] = blockIdx.x / _del[idx] % 2;
	bin_dev[bli+idx+32768*16] = bin_dev[bli+idx];
	bin_dev[bli+15+32768*16] = 1;
}


__global__ void smth(int *bin_dev, int* wight_dev, int *s_dev, int*values_dev)
{					//bin_dev,wight_dev,s_dev,values_dev
	int bli = blockIdx.x * blockDim.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	s_dev[i] = bin_dev[i] * values_dev[threadIdx.x];//s_dev -> prices*bin
	s_dev[i+32768*16] = s_dev[i];
	s_dev[bli+15+32768*16] = values_dev[15];
	bin_dev[i] = bin_dev[i] * wight_dev[threadIdx.x];//bin_dev -> weights*bin
	bin_dev[i+32768*16] = bin_dev[i];
	bin_dev[bli+15+32768*16] = wight_dev[15];
}

__global__ void plusing2_w(int* in_dev, int* sums) {
	__shared__ int sdata[16];
	// each thread loads one element from global to shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	sdata[tid] = in_dev[i];
	__syncthreads();
	//unroled summing is faster than reduction according to my tests

sdata[0] += sdata[1];
sdata[0] += sdata[2];
sdata[0] += sdata[3];
sdata[0] += sdata[4];
sdata[0] += sdata[5];
sdata[0] += sdata[6];
sdata[0] += sdata[7];
sdata[0] += sdata[8];
sdata[0] += sdata[9];
sdata[0] += sdata[10];
sdata[0] += sdata[11];
sdata[0] += sdata[12];
sdata[0] += sdata[13];
sdata[0] += sdata[14];
//sdata[0] += sdata[15];

		__syncthreads();

	// write result for this block to global mem
	if (tid == 0) sums[blockIdx.x] = sdata[0];
}
__global__ void plusing2_w2(int* add, int* sums) {

	// each thread loads one element from global to shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	sums[i+32768]=sums[i]+add[15];
}


__global__ void zeroing(int *w, int *s, int W) {
	int bli = blockIdx.x * blockDim.x;
	int idx = threadIdx.x;
	if (w[bli+idx] > W) { s[bli + idx] = 0; w[bli + idx] = 0; }
}
__global__ void max1(int* s) {
	__shared__ int sdata[1024];
	// each thread loads one element from global to shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	sdata[tid] = s[i];
	__syncthreads();
	// do reduction in shared mem
	for (unsigned int s = blockDim.x / 2; s>0; s >>= 1) {
		if (tid < s) {
			if (sdata[tid] < sdata[tid + s])
				sdata[tid] = sdata[tid + s];
		}
		__syncthreads();
	}
	// write result for this block to global mem
	if (tid == 0) s[blockIdx.x] = sdata[0];
}

__global__ void kermax2(int *s, int N) {
	//__shared__ int max[32];
	unsigned int tid = threadIdx.x;
	int off = N / 2;
	if (tid < off) {
		if (s[tid] < s[tid + off]) {
			s[tid] = s[tid + off];
		}
	}
}

int main()
{
	int qwerty = 0;
	while (qwerty < 1)
	{

		int W = 300;
		int arraySize = 16;
		int arraySize_b = 32768*2 * 16;
		int Sum[65536*16] = { 0 };
		int *s;
		int *bin_dev;
		int *wight_dev;
		int wight[16] = { 5,10,17,19,20, 23,26,30,32,38,   40,44,47,50,55 ,56};// 55, 56, 60, 62, 66, 70	};
		int values[16] = { 10,13,16,22,30, 25,55,90,110,115, 130,120,150,170,194,199 };// , 194, 199, 217, 230, 248	};
		int *w;
		int *values_dev;


		int del[16], *_del;
		hipMalloc((void**)&_del, 16 * sizeof(int));
		for (int i = 0; i < 16; i++) {
			del[i] = pow(2, i);cout<<del[i]<<" ";
		}

		float gpu_elapsed_time,sumoftime=0;

		hipEvent_t gpu_start, gpu_stop;
		hipEventCreate(&gpu_start);
		hipEventCreate(&gpu_stop);
		hipEventRecord(gpu_start, 0);

		hipMemcpy(_del, del, 16 * sizeof(int), hipMemcpyHostToDevice);
		hipMalloc((void**)&bin_dev, arraySize_b * sizeof(int));

		int*s_dev;
		hipMalloc((void**)&s_dev, arraySize_b * sizeof(int));
		hipMalloc((void**)&wight_dev, arraySize * sizeof(int));
		hipMalloc((void**)&s, arraySize_b * sizeof(int));
		hipMalloc((void**)&values_dev, arraySize * sizeof(int));
		hipMalloc((void**)&w, 32768 *2* sizeof(int));
		hipMemcpy(wight_dev, wight, arraySize * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(values_dev, values, arraySize * sizeof(int), hipMemcpyHostToDevice);

		hipEventRecord(gpu_stop, 0);
		hipEventSynchronize(gpu_stop);
		hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
		hipEventDestroy(gpu_start);
		hipEventDestroy(gpu_stop);
sumoftime+=gpu_elapsed_time;

hipEventCreate(&gpu_start);
hipEventCreate(&gpu_stop);
hipEventRecord(gpu_start, 0);

		cout << "Copying all needed data to GPU: " << gpu_elapsed_time << " milli-seconds\n";
sumoftime+=gpu_elapsed_time;
gpu_elapsed_time = 0;
		hipEventCreate(&gpu_start);
		hipEventCreate(&gpu_stop);
		hipEventRecord(gpu_start, 0);

		T_binary3 << <32768, 16 >> > (bin_dev, _del);
/*
hipMemcpy(Sum, bin_dev, 32768*2*16*sizeof(int), hipMemcpyDeviceToHost);
for(int i=0;i<5;i++){
	for(int j=0;j<16;j++){cout<<Sum[i*16+j];}cout<<"\n";
}cout<<"---------------\n";
for(int i=32763;i<32768;i++){
	for(int j=0;j<16;j++){cout<<Sum[i*16+j];}cout<<"\n";
}cout<<"---------------\n";
for(int i=32768;i<32773;i++){
	for(int j=0;j<16;j++){cout<<Sum[i*16+j];}cout<<"\n";
}
cout<<"---------------\n";
for(int i=65530;i<65536;i++){
	for(int j=0;j<16;j++){cout<<Sum[i*16+j];}cout<<"\n";
}
*/

		hipEventRecord(gpu_stop, 0);
		hipEventSynchronize(gpu_stop);
		hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
		hipEventDestroy(gpu_start);
		hipEventDestroy(gpu_stop);
		cout << "T_binary3: " << gpu_elapsed_time << " milli-seconds\n";
sumoftime+=gpu_elapsed_time;
gpu_elapsed_time = 0;


		hipEventCreate(&gpu_start);
		hipEventCreate(&gpu_stop);
		hipEventRecord(gpu_start, 0);

		smth << <32768,16 >> > (bin_dev, wight_dev, s_dev, values_dev);

		hipEventRecord(gpu_stop, 0);
		hipEventSynchronize(gpu_stop);
		hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
		hipEventDestroy(gpu_start);
		hipEventDestroy(gpu_stop);
		cout << "smth: " << gpu_elapsed_time << " milli-seconds\n";
sumoftime+=gpu_elapsed_time;
gpu_elapsed_time = 0;
/*
cout<<"-|-|-|-|-|-|-|-|-|-\n";
for(int j=0;j<16;j++){cout<<values[j]<<" ";}cout<<"\n";
cout<<"-|-|-|-|-|-|-|-|-|-\n";
hipMemcpy(Sum, s_dev, 32768*2*16*sizeof(int), hipMemcpyDeviceToHost);
for(int i=0;i<5;i++){
	for(int j=0;j<16;j++){cout<<Sum[i*16+j]<<" ";}cout<<"\n";
}cout<<"---------------\n";
for(int i=32763;i<32768;i++){
	for(int j=0;j<16;j++){cout<<Sum[i*16+j]<<" ";}cout<<"\n";
}cout<<"---------------\n";
for(int i=32768;i<32773;i++){
	for(int j=0;j<16;j++){cout<<Sum[i*16+j]<<" ";}cout<<"\n";
}
cout<<"---------------\n";
for(int i=65530;i<65536;i++){
	for(int j=0;j<16;j++){cout<<Sum[i*16+j]<<" ";}cout<<"\n";
}

*/

		hipEventCreate(&gpu_start);
		hipEventCreate(&gpu_stop);
		hipEventRecord(gpu_start, 0);

		plusing2_w << <32768, 16 >> > (bin_dev, w);
		plusing2_w2 << <32,1024 >> > (wight_dev,w);
		hipEventRecord(gpu_stop, 0);
		hipEventSynchronize(gpu_stop);
		hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
		hipEventDestroy(gpu_start);
		hipEventDestroy(gpu_stop);
		cout << "plusing_weight: " << gpu_elapsed_time << " milli-seconds\n";
sumoftime+=gpu_elapsed_time;
gpu_elapsed_time = 0;

/*
hipMemcpy(Sum, w, 65536*sizeof(int), hipMemcpyDeviceToHost);
for(int i=0;i<65536;i++){cout<<Sum[i]<<" ";}
*/

		hipEventCreate(&gpu_start);
		hipEventCreate(&gpu_stop);
		hipEventRecord(gpu_start, 0);

		plusing2_w << <32768, 16 >> > (s_dev, s);
	  plusing2_w2 << <32,1024 >> > (values_dev,s);
		hipEventRecord(gpu_stop, 0);
		hipEventSynchronize(gpu_stop);
		hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
		hipEventDestroy(gpu_start);
		hipEventDestroy(gpu_stop);
		cout << "plusing_prices: " << gpu_elapsed_time << " milli-seconds\n";
sumoftime+=gpu_elapsed_time;
gpu_elapsed_time = 0;

/*
hipMemcpy(Sum, s, 65536*sizeof(int), hipMemcpyDeviceToHost);
for(int i=0;i<65536;i++){cout<<Sum[i]<<" ";}
*/
		hipEventCreate(&gpu_start);
		hipEventCreate(&gpu_stop);
		hipEventRecord(gpu_start, 0);

		zeroing << <64, 1024 >> > (w, s, W);

		hipEventRecord(gpu_stop, 0);
		hipEventSynchronize(gpu_stop);
		hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
		hipEventDestroy(gpu_start);
		hipEventDestroy(gpu_stop);
		cout << "zeroing: " << gpu_elapsed_time << " milli-seconds\n";
sumoftime+=gpu_elapsed_time;
gpu_elapsed_time = 0;

/*
hipMemcpy(Sum, s, 65536*sizeof(int), hipMemcpyDeviceToHost);
for(int i=0;i<65536;i++){cout<<Sum[i]<<" ";}
*/

		hipEventCreate(&gpu_start);
		hipEventCreate(&gpu_stop);
		hipEventRecord(gpu_start, 0);

		max1 << <64, 1024 >> > (s);

		hipEventRecord(gpu_stop, 0);
		hipEventSynchronize(gpu_stop);
		hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
		hipEventDestroy(gpu_start);
		hipEventDestroy(gpu_stop);
		cout << "max1: " << gpu_elapsed_time << " milli-seconds\n";
sumoftime+=gpu_elapsed_time;
gpu_elapsed_time = 0;

/*
hipMemcpy(Sum, s, 64*sizeof(int), hipMemcpyDeviceToHost);
for(int i=0;i<64;i++){cout<<Sum[i]<<" ";}
*/


		hipEventCreate(&gpu_start);
		hipEventCreate(&gpu_stop);
		hipEventRecord(gpu_start, 0);
kermax2 << <2, 32 >> > (s,32);
		for (int i = 32; i >= 1; i /= 2) {
			kermax2 << <1, i >> > (s,i);
		}

		hipEventRecord(gpu_stop, 0);
		hipEventSynchronize(gpu_stop);
		hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
		hipEventDestroy(gpu_start);
		hipEventDestroy(gpu_stop);
		cout << "loops of kermax2: " << gpu_elapsed_time << " milli-seconds\n";
sumoftime+=gpu_elapsed_time;
gpu_elapsed_time = 0;


		hipEventCreate(&gpu_start);
		hipEventCreate(&gpu_stop);
		hipEventRecord(gpu_start, 0);

		hipMemcpy(Sum, s, sizeof(int), hipMemcpyDeviceToHost);

		hipEventRecord(gpu_stop, 0);
		hipEventSynchronize(gpu_stop);
		hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
		hipEventDestroy(gpu_start);
		hipEventDestroy(gpu_stop);
		cout << "cpy of maximal sum: " << gpu_elapsed_time << " milli-seconds\n";
sumoftime+=gpu_elapsed_time;
gpu_elapsed_time = 0;
		cout <<"\n"<<"max = " << Sum[0] << "\n";


		cout << "Total time: " << sumoftime << " milli-seconds\n";

		hipFree(bin_dev);
		hipFree(wight_dev);
		hipFree(s);
		hipFree(w);
		hipFree(s_dev);
		cout << "\n";


		//system("pause");

		//CPU version 1
		float fTimeStart = clock() / (float)(CLOCKS_PER_SEC);
		//int bin[32768*16] = { 0 };
		int k = 0;
		for (int i = 0; i < 32768; i++) {
			k = i;
			for (int j = 15; j >= 0; j--) {
				Sum[i*16+j] = k % 2;
				k /= 2;
			}
		}

		//int wight[15] = { 5,10,17,19,20, 23,26,30,32,38,   40,44,47,50,55 };// 55, 56, 60, 62, 66, 70	};
		//int values[15] = { 10,13,16,22,30, 25,55,90,110,115, 130,120,150,170,194 };// , 194, 199, 217, 230, 248	};
		//int prices[32768][16] = { 0 };
		//int weights[32768][16] = { 0 };
		int Sweig[65536] = { 0 };
		int Sval[65536] = { 0 };


		for (int i = 0; i < 32768; i++) {
			for (int j = 0; j < 16; j++) {
				Sum[i*16+j] = wight[j] * Sum[i*16+j];
				//prices[i][j] = values[j] * Sum[i*16+j];
			}
		}

		for (int i = 0; i < 32768; i++) {
			for (int j = 0; j < 16; j++) {
				Sweig[i] +=Sum[i*16+j];;
				//Sval[i] += prices[i][j];
			}
		}
		for (int i = 32768; i < 65536; i++) {
			for (int j = 0; j < 16; j++) {
				Sweig[i] =Sweig[i-32768]+ wight[15];
				//Sval[i] =Sval[i]+values[15];
			}
		}
		for (int i = 0; i < 32768; i++) {
			for (int j = 0; j < 16; j++) {
//			Sum[i*16+j] = wight[j] * Sum[i*16+j];
				Sum[i*16+j] = values[j] * Sum[i*16+j];
			}
		}
		for (int i = 32768; i < 65536; i++) {
			for (int j = 0; j < 16; j++) {
				//Sweig[i] =Sweig[i-32768]+ wight[15];
				Sval[i] =Sval[i-32768]+values[15];
			}
		}

		int max = 0; k = 0;
		for (int i = 0; i < 65536; i++) {
			if ((Sweig[i] <= W) && (Sval[i] > max)) {
				k = i; max = Sval[i];
			}
		}
		float fTimeStop = clock() / (float)CLOCKS_PER_SEC;
		cout << "\nmax = " << max << "\n" << "String No " << k << "\n";
		//for (int i = 0; i < 10; i++) { cout << bin[k][i] << " "; }cout << "\n";
		//for (int i = 0; i < 10; i++) { if (weights[k][i] != 0)cout << weights[k][i] << " + "; }cout << "\n";
		cout << "CPU time is " << (fTimeStop - fTimeStart) * 1000 << " milli-seconds\n";



		qwerty++;
	}


	return 0;
}

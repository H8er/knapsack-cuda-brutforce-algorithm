
#include "hip/hip_runtime.h"

#include "hip/device_functions.h"
#include <cstdlib>
#include <stdlib.h>
#include <sys/time.h>
#include <iostream>

#define arraySize 30
#define threads_per_block 1024
#define max_blocks 32768
using namespace std;


__global__ void single_thread(int *sh_sum_dev, int *weight_dev, int *values_dev, int W, int *str_num_dev, int num_of_blocks,int rep)
{
  int th_w_sum = 0;
  int th_v_sum = 0;
  int th_bin[arraySize];
  __shared__ int sh_w_d[arraySize];
  __shared__ int sh_v_d[arraySize];
  __shared__ int sh_maxs[threads_per_block];
  __shared__ int indices[threads_per_block];
  indices[threadIdx.x] = threadIdx.x;

if(threadIdx.x<arraySize){
	sh_w_d[threadIdx.x] = weight_dev[threadIdx.x];
	sh_v_d[threadIdx.x] = values_dev[threadIdx.x];
}

__syncthreads();
int num_to_bin = blockIdx.x * blockDim.x + threadIdx.x;
num_to_bin += max_blocks * rep;
#pragma unroll
  for (uint i = 0; i < arraySize; i++)
    {
      th_bin[i] = ((num_to_bin) >> i) % 2;
      th_w_sum += th_bin[i] * sh_w_d[i];
      th_v_sum += th_bin[i] * sh_v_d[i];
    }

sh_maxs[threadIdx.x] = (th_w_sum > W) ? 0:th_v_sum;


__syncthreads ();
  for (unsigned int offset = blockDim.x / 2; offset >= 1; offset >>= 1)
    {
      if (threadIdx.x < offset)
	{
	  if (sh_maxs[threadIdx.x] < sh_maxs[threadIdx.x + offset])
	    {
	      sh_maxs[threadIdx.x] = sh_maxs[threadIdx.x + offset];
	      indices[threadIdx.x] = indices[threadIdx.x + offset];
	    }
	}
      __syncthreads ();
    }
  // write result for this block to global mem
  sh_sum_dev[blockIdx.x+max_blocks*rep] = sh_maxs[0];
  //sh_sum_dev[blockIdx.x + num_of_blocks] = sh_we[0];
  str_num_dev[blockIdx.x+max_blocks*rep] = indices[0] + blockIdx.x * blockDim.x +max_blocks*rep;
}

__global__ void
reduction_max (int *s, int *str_num_dev)
{
  int ID = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ int sdata[threads_per_block*2];
  sdata[threadIdx.x] = s[ID];
  sdata[threadIdx.x + threads_per_block] = str_num_dev[ID];

  __syncthreads ();
  // do reduction in shared mem
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
      if (threadIdx.x < s)
	{
	  if (sdata[threadIdx.x] < sdata[threadIdx.x + s])
	    {
	      sdata[threadIdx.x] = sdata[threadIdx.x + s];
	      //sdata[tid+64] = sdata[tid + s+64];
	      sdata[threadIdx.x + threads_per_block] =
		sdata[threadIdx.x + threads_per_block + s];
	    }
	}
      __syncthreads ();
    }
  // write result for this block to global mem
  if (threadIdx.x == 0)
    {
			//if(sdata[0]>s[0]){//}&&(blockIdx.x>0)){
      s[blockIdx.x] = sdata[0];
      str_num_dev[blockIdx.x] = sdata[threads_per_block];
		}
    //}
}

__global__ void
which_string (int a, int *view_dev)
{
  view_dev[threadIdx.x] = (a >> threadIdx.x) % 2;
}

int main(){
  int W = 500;

  struct timeval t0, t1;

  int strSize_b = pow (2, arraySize);
  int num_of_blocks = strSize_b / threads_per_block;
  int *Sum = new int[1];	// = { 0 };
  int *sh_sum_dev;
  int *weight_dev;
  int weight[30] ={ 5, 10, 17, 19, 20, 23, 26, 30, 32, 38, 40, 44, 47, 50, 55, 56, 56, 60, 62, 66, 70, 75, 77, 80, 81, 90,93,96,101,107 };
  int values[30] ={ 10, 13, 16, 22, 30, 25, 55, 90, 110, 115, 130, 120, 150, 170, 194, 199, 194, 199, 217, 230, 248, 250, 264, 271, 279, 286,293,299,305,313 };
  int *values_dev;
  long sec, usec;
  int *str_num_dev;
  int *str_num = new int[1];

  cout <<"N of items "<<arraySize<<"\n";
  cout<<"N of blocks "<<num_of_blocks<<"\n";
  cout<<"strSize_b = "<<strSize_b<<"\n";
  cout<<"num_of_blocks / threads_per_block = "<<num_of_blocks / threads_per_block<<"\n";
  cout<<"IDEA  "<<strSize_b/num_of_blocks<<"\n";
  cout<<"red param "<<1 + (num_of_blocks-1) / threads_per_block<<"  ,  "<<strSize_b/num_of_blocks<<"\n";

  int N_of_rep;
  N_of_rep = num_of_blocks/max_blocks;
  cout<<"N of repeats = "<<N_of_rep<<"\n";
  cout<<"sing param = "<<num_of_blocks/N_of_rep<<" _ "<< threads_per_block<<"\n";
  gettimeofday (&t0, NULL);
  hipMalloc ((void **) &sh_sum_dev,  num_of_blocks * sizeof (int));
  hipMalloc ((void **) &weight_dev, arraySize * sizeof (int));
  hipMalloc ((void **) &values_dev, arraySize * sizeof (int));
  hipMalloc ((void **) &str_num_dev, num_of_blocks * sizeof (int));

  hipMemcpy (weight_dev, weight, arraySize * sizeof (int),
	      hipMemcpyHostToDevice);
  hipMemcpy (values_dev, values, arraySize * sizeof (int),
	      hipMemcpyHostToDevice);


        for(int i = 0;i<N_of_rep;i++){
  single_thread <<< num_of_blocks/N_of_rep, threads_per_block >>> (sh_sum_dev, weight_dev, values_dev,
					     W, str_num_dev, num_of_blocks,i);
             }
//cout all the parameters


  reduction_max <<<1 + (num_of_blocks-1) / threads_per_block, strSize_b/num_of_blocks>>> (sh_sum_dev, str_num_dev);
  reduction_max <<<1, 1 + (num_of_blocks-1) / threads_per_block>>> (sh_sum_dev, str_num_dev);

  hipMemcpy (Sum, sh_sum_dev, sizeof (int), hipMemcpyDeviceToHost);
  hipMemcpy (str_num, str_num_dev, sizeof (int), hipMemcpyDeviceToHost);
	gettimeofday (&t1, 0);
	sec = (t1.tv_sec - t0.tv_sec);
  usec = t1.tv_usec - t0.tv_usec;
  cout << "GPU time = " << sec << " sec, " << usec << " microsec\n";

  cout << "Acheived maximal sum = " << Sum[0] << "\n";
  cout << "String number " << str_num[0] << "\n";

  int *view = new int[arraySize];
  int *view_dev;
  hipMalloc ((void **) &view_dev, arraySize * sizeof (int));
  which_string <<< 1, arraySize >>> (str_num[0], view_dev);
  hipMemcpy (view, view_dev, arraySize * sizeof (int),
	      hipMemcpyDeviceToHost);
  for (int i = 0; i < arraySize; i++)
    {
      cout << view[i] << " ";
    } cout << "\n";
  //check
  int checksum = 0;
  for (int i = 0; i < arraySize; i++)
    {
      checksum += values[i] * view[i];
    } cout << "Validation sum = " << checksum << "\n";
  checksum = 0;
  for (int i = 0; i < arraySize; i++)
    {
      checksum += weight[i] * view[i];
    } cout << "Weight = " << checksum << "\n";
  hipFree (sh_sum_dev);
  hipFree (weight_dev);
  hipFree (values_dev);
  hipFree (view_dev);

  return 0;
}

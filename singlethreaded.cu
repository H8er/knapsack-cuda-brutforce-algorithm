
#include "hip/hip_runtime.h"

#include "hip/device_functions.h"
#include <cstdlib>
#include <stdlib.h>
#include <sys/time.h>
#include <iostream>

#define arraySize 16
using namespace std;

__global__ void single_thread(int* bin_dev,int* weight_dev, int*values_dev,int W,int* NO){
	int th_w_sum = 0;
	int th_v_sum = 0;
	int th_bin[arraySize];
	__shared__ int sh_w_d[arraySize];
	__shared__ int sh_v_d[arraySize];
	__shared__ int sh_maxs[1024];
	__shared__ int sh_we[1024];

	#pragma unroll
	for(uint i = 0;i<arraySize;i++){
		th_bin[i] = ((blockIdx.x*blockDim.x + threadIdx.x)>>i)%2;
		sh_w_d[i] = weight_dev[i];
		sh_v_d[i] = values_dev[i];
		th_w_sum+=th_bin[i]*sh_w_d[i];
		th_v_sum+=th_bin[i]*sh_v_d[i];
	}

	sh_maxs[threadIdx.x] = th_v_sum;
	sh_we[threadIdx.x] = th_w_sum;
	__syncthreads();
	if(sh_we[threadIdx.x]>W){sh_maxs[threadIdx.x]=0;sh_we[threadIdx.x]=0;
	}

	for (unsigned int s = blockDim.x / 2; s>=1; s >>= 1) {
		if(threadIdx.x<s){
			if(sh_maxs[threadIdx.x]<sh_maxs[threadIdx.x + s]){
			sh_maxs[threadIdx.x]=sh_maxs[threadIdx.x + s];
			sh_we[threadIdx.x] = sh_we[threadIdx.x + s];
		}}
		__syncthreads();
	}
	// write result for this block to global mem
bin_dev[blockIdx.x] = sh_maxs[0];
bin_dev[blockIdx.x+64] = sh_we[0];
}

__global__ void reduction_max(int* s) {
	extern __shared__ int sdata[];
	// each thread loads one element from global to shared mem
	unsigned int tid = threadIdx.x;
	//unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	sdata[tid] = s[tid];
	__syncthreads();
	// do reduction in shared mem
	for (unsigned int s = blockDim.x / 2; s>0; s >>= 1) {
		if (tid < s) {
			if (sdata[tid] < sdata[tid + s]){
				sdata[tid] = sdata[tid + s];
				//sdata[tid+64] = sdata[tid + s+64];
		}}
		__syncthreads();
	}
	// write result for this block to global mem
	if (tid == 0) {s[0] =sdata[0];}//;s[1] = sdata[65];}
}

int main()
{
		int W = 350;

		struct timeval t0,t1;

		int totalSize = arraySize*pow(2,arraySize);
		int strSize_b = pow(2,arraySize);

		int *Sum=new int[strSize_b/1024*2];// = { 0 };
		int *bin_dev;
		int *weight_dev;
		int weight[16] = { 5,10,17,19,20, 23,26,30,32,38, 40,44,47,50,55,56 };// 55, 56, 60, 62, 66, 70	};
		int values[16] = { 10,13,16,22,30, 25,55,90,110,115, 130,120,150,170,194,199 };// , 194, 199, 217, 230, 248	};
		int *values_dev;

		hipMalloc((void**)&bin_dev, 2*strSize_b/1024 * sizeof(int));
		hipMalloc((void**)&weight_dev, arraySize * sizeof(int));
		hipMalloc((void**)&values_dev, arraySize * sizeof(int));

		hipMemcpy(weight_dev, weight, arraySize * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(values_dev, values, arraySize * sizeof(int), hipMemcpyHostToDevice);


long sec,usec;

		int *w_sum;
		int *v_sum;
		int *NO;
		hipMalloc((void**)&w_sum, strSize_b/1024 * sizeof(int));
		hipMalloc((void**)&v_sum, strSize_b/1024 * sizeof(int));
		hipMalloc((void**)&NO, sizeof(int));

gettimeofday(&t0, NULL);
single_thread<<< strSize_b/1024,1024>>>(bin_dev,weight_dev,values_dev,W,NO);
reduction_max<<<1,strSize_b/1024,strSize_b/1024*sizeof(int)>>>(bin_dev);
gettimeofday(&t1, 0);
hipMemcpy(Sum, bin_dev, 2*sizeof(int), hipMemcpyDeviceToHost);
sec = (t1.tv_sec-t0.tv_sec);
usec =  t1.tv_usec-t0.tv_usec;
cout<<"GPU time = "<<sec<<" sec, "<<usec<<" microsec\n";

cout<<"Acheived maximal sum = "<<Sum[0]<<"\n";


		hipFree(bin_dev);
		hipFree(weight_dev);
    hipFree(values_dev);
		hipFree(NO);

return 0;
}

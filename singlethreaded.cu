
		#include "hip/hip_runtime.h"
		
		#include "hip/device_functions.h"
		#include <cstdlib>
		#include <stdlib.h>
		#include <sys/time.h>
		#include <iostream>

		#define arraySize 20
		using namespace std;

		__global__ void single_thread(int* bin_dev,int* weight_dev, int*values_dev,int W,int* str_num_dev,int num_of_blocks){
			int th_w_sum = 0;
			int th_v_sum = 0;
			int th_bin[arraySize];
			__shared__ int sh_w_d[arraySize];
			__shared__ int sh_v_d[arraySize];
			__shared__ int sh_maxs[1024];
			__shared__ int sh_we[1024];
			__shared__ int indices[1024];
			indices[threadIdx.x] = threadIdx.x;
			#pragma unroll
			for(uint i = 0;i<arraySize;i++){
				th_bin[i] = ((blockIdx.x*blockDim.x + threadIdx.x)>>i)%2;
				sh_w_d[i] = weight_dev[i];
				sh_v_d[i] = values_dev[i];
				th_w_sum+=th_bin[i]*sh_w_d[i];
				th_v_sum+=th_bin[i]*sh_v_d[i];
			}

			sh_maxs[threadIdx.x] = th_v_sum;
			sh_we[threadIdx.x] = th_w_sum;
			__syncthreads();
			if(sh_we[threadIdx.x]>W){sh_maxs[threadIdx.x]=0;sh_we[threadIdx.x]=0;
			}

			for (unsigned int s = blockDim.x / 2; s>=1; s >>= 1) {
				if(threadIdx.x<s){
					if(sh_maxs[threadIdx.x]<sh_maxs[threadIdx.x + s]){
					sh_maxs[threadIdx.x]=sh_maxs[threadIdx.x + s];
					sh_we[threadIdx.x] = sh_we[threadIdx.x + s];
					indices[threadIdx.x] = indices[threadIdx.x + s];
				}}
				__syncthreads();
			}
			// write result for this block to global mem
		bin_dev[blockIdx.x] = sh_maxs[0];
		bin_dev[blockIdx.x+num_of_blocks] = sh_we[0];
		str_num_dev[blockIdx.x] = indices[0]+blockIdx.x*blockDim.x;
		}

		__global__ void reduction_max(int* s,int* str_num_dev,int num_of_blocks) {
			extern __shared__ int sdata[];
			sdata[threadIdx.x+num_of_blocks] = str_num_dev[threadIdx.x];
			//red_ind[threadIdx.x] = str_num_dev[threadIdx.x];
			// each thread loads one element from global to shared mem
			//unsigned int tid = threadIdx.x;
			//unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
			sdata[threadIdx.x] = s[threadIdx.x];
			__syncthreads();
			// do reduction in shared mem
			for (unsigned int s = blockDim.x / 2; s>0; s >>= 1) {
				if (threadIdx.x < s) {
					if (sdata[threadIdx.x] < sdata[threadIdx.x + s]){
						sdata[threadIdx.x] = sdata[threadIdx.x + s];
						//sdata[tid+64] = sdata[tid + s+64];
						sdata[threadIdx.x+num_of_blocks] = sdata[threadIdx.x+num_of_blocks + s];
				}}
				__syncthreads();
			}
			// write result for this block to global mem
			if (threadIdx.x == 0) {s[0] =sdata[0]; str_num_dev[0] = sdata[num_of_blocks];}//;s[1] = sdata[65];}
		}


		__global__ void which_string(int a,int*view_dev){
				view_dev[threadIdx.x] = (a>>threadIdx.x)%2;
		}
		int main()
		{
				int W = 350;

				struct timeval t0,t1;

				int totalSize = arraySize*pow(2,arraySize);
				int strSize_b = pow(2,arraySize);
				int num_of_blocks = strSize_b/1024;
				int *Sum=new int[num_of_blocks*2];// = { 0 };
				int *bin_dev;
				int *weight_dev;
				int weight[21] = { 5,10,17,19,20, 23,26,30,32,38, 40,44,47,50,55, 56,56,60,62, 66, 70	};
				int values[21] = { 10,13,16,22,30, 25,55,90,110,115, 130,120,150,170,194,199, 194, 199, 217, 230, 248	};
				int *values_dev;

				hipMalloc((void**)&bin_dev, 2*num_of_blocks * sizeof(int));
				hipMalloc((void**)&weight_dev, arraySize * sizeof(int));
				hipMalloc((void**)&values_dev, arraySize * sizeof(int));

				int *str_num_dev;
				int*str_num = new int[num_of_blocks];
				hipMalloc((void**)&str_num_dev,num_of_blocks*sizeof(int));

				hipMemcpy(weight_dev, weight, arraySize * sizeof(int), hipMemcpyHostToDevice);
				hipMemcpy(values_dev, values, arraySize * sizeof(int), hipMemcpyHostToDevice);


		long sec,usec;

				int *w_sum;
				int *v_sum;

				hipMalloc((void**)&w_sum, num_of_blocks * sizeof(int));
				hipMalloc((void**)&v_sum, num_of_blocks * sizeof(int));


		gettimeofday(&t0, NULL);

		single_thread<<< num_of_blocks,1024>>>(bin_dev,weight_dev,values_dev,W,str_num_dev,num_of_blocks);
		reduction_max<<<1,num_of_blocks,num_of_blocks*sizeof(int)*2>>>(bin_dev,str_num_dev,num_of_blocks);

		gettimeofday(&t1, 0);
		hipMemcpy(Sum, bin_dev, 2*sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(str_num, str_num_dev, 1*sizeof(int), hipMemcpyDeviceToHost);
		sec = (t1.tv_sec-t0.tv_sec);
		usec =  t1.tv_usec-t0.tv_usec;
		cout<<"GPU time = "<<sec<<" sec, "<<usec<<" microsec\n";

		cout<<"Acheived maximal sum = "<<Sum[0]<<"\n";
		cout<<"String number "<<str_num[0]<<"\n";
		hipMemcpy(str_num, str_num_dev, 1*sizeof(int), hipMemcpyDeviceToHost);
		int*view = new int[arraySize];
		int*view_dev;
		hipMalloc((void**)&view_dev, arraySize*sizeof(int));
		which_string<<<1,arraySize>>>(str_num[0],view_dev);
		hipMemcpy(view, view_dev, arraySize*sizeof(int), hipMemcpyDeviceToHost);
		for(int i = 0;i<arraySize;i++){
			cout<<view[i]<<" ";
		}cout<<"\n";
		//check
		int checksum = 0;
		for(int i = 0;i<arraySize;i++){
			checksum+=values[i]*view[i];
		}cout<<"Validation sum = "<<checksum<<"\n";
		checksum = 0;
		for(int i = 0;i<arraySize;i++){
			checksum+=weight[i]*view[i];
		}cout<<"Weight = "<<checksum<<"\n";
				hipFree(bin_dev);
				hipFree(weight_dev);
		    hipFree(values_dev);
				hipFree(view_dev);

		return 0;
		}

#include "hip/hip_runtime.h"

//Compile in MSVS15 as it is
//Compile in Linux-systems using "nvcc -ccbin clang kernel.cu"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <cstdlib>
#include <stdlib.h>
#include <ctime>
#include <time.h>
#include <iostream>


using namespace std;

//bin_dev, wight_dev, s_dev, values_dev);
__global__ void smth(int *bin_dev, int* wight_dev, int *s_dev, int*values_dev)
{					//bin_dev,wight_dev,s_dev,values_dev
	int i = blockIdx.x * blockDim.x + threadIdx.x;


	s_dev[i] = bin_dev[i] * values_dev[threadIdx.x];//s_dev -> prices*bin
	bin_dev[i] = bin_dev[i] * wight_dev[threadIdx.x];//bin_dev -> weights*bin
}


__global__ void kermax2(int *s, int N) {
	//__shared__ int max[32];
	unsigned int tid = threadIdx.x;
	int off = N / 2;
	if (tid < off) {
		if (s[tid] < s[tid + off]) {
			s[tid] = s[tid + off];
		}
	}
}


__global__ void T_binary3(int*bin_dev, int *_del) {
	int bli = blockIdx.x * blockDim.x;
	int idx = threadIdx.x;
	bin_dev[bli + idx] = blockIdx.x / _del[idx] % 2;
}
__global__ void summing(int *s_dev, int*w, int*s, int*bin_dev, int W) {
	int bli = blockIdx.x * blockDim.x;
	for (int i = 0; i < 15; i++) {
		w[blockIdx.x] += bin_dev[bli + i];
		s[blockIdx.x] += s_dev[bli + i];
	}
	if (w[blockIdx.x] > W) { s[blockIdx.x] = 0; w[blockIdx.x] = 0; }
	//15x32x1024

}

__global__ void plusing2_w(int* in_dev, int* sums) {
	__shared__ int sdata[15];
	// each thread loads one element from global to shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	sdata[tid] = in_dev[i];
	__syncthreads();
	//unroled summing is faster than reduction according to my tests

sdata[0] += sdata[1];
sdata[0] += sdata[2];
sdata[0] += sdata[3];
sdata[0] += sdata[4];
sdata[0] += sdata[5];
sdata[0] += sdata[6];
sdata[0] += sdata[7];
sdata[0] += sdata[8];
sdata[0] += sdata[9];
sdata[0] += sdata[10];
sdata[0] += sdata[11];
sdata[0] += sdata[12];
sdata[0] += sdata[13];
sdata[0] += sdata[14];

		__syncthreads();
	
	// write result for this block to global mem
	if (tid == 0) sums[blockIdx.x] = sdata[0];
}

__global__ void zeroing(int *w, int *s, int W) {
	int bli = blockIdx.x * blockDim.x;
	int idx = threadIdx.x;
	if (w[bli+idx] > W) { s[bli + idx] = 0; w[bli + idx] = 0; }
}
__global__ void max1(int* s) {
	__shared__ int sdata[1024];
	// each thread loads one element from global to shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	sdata[tid] = s[i];
	__syncthreads();
	// do reduction in shared mem
	for (unsigned int s = blockDim.x / 2; s>0; s >>= 1) {
		if (tid < s) {
			if (sdata[tid] < sdata[tid + s])
				sdata[tid] = sdata[tid + s];
		}
		__syncthreads();
	}
	// write result for this block to global mem
	if (tid == 0) s[blockIdx.x] = sdata[0];
}

int main()
{
	int qwerty = 0;
	while (qwerty < 1)
	{

		int W = 300;
		int arraySize = 15;
		int arraySize_b = 32768 * 15;
		int Sum[32768 * 15] = { 0 };
		int *s;
		//int bin[32768 * 15];
		int *bin_dev;
		int *wight_dev;
		//int prices[32768 * 15] = { 0 };
		int wight[15] = { 5,10,17,19,20, 23,26,30,32,38,   40,44,47,50,55 };// 55, 56, 60, 62, 66, 70	};
		int values[15] = { 10,13,16,22,30, 25,55,90,110,115, 130,120,150,170,194 };// , 194, 199, 217, 230, 248	};
		int *w;
		int *values_dev;


		int del[15], *_del;
		hipMalloc((void**)&_del, 15 * sizeof(int));
		for (int i = 0; i < 15; i++) {
			del[i] = pow(2, i);
		}

		float gpu_elapsed_time,sumoftime=0;
/*
		hipEvent_t gpu_start, gpu_stop;
		hipEventCreate(&gpu_start);
		hipEventCreate(&gpu_stop);
		hipEventRecord(gpu_start, 0);
*/
		hipMemcpy(_del, del, 15 * sizeof(int), hipMemcpyHostToDevice);
		hipMalloc((void**)&bin_dev, arraySize_b * sizeof(int));

		int*s_dev;
		hipMalloc((void**)&s_dev, arraySize_b * sizeof(int));
		hipMalloc((void**)&wight_dev, arraySize * sizeof(int));
		hipMalloc((void**)&s, arraySize_b * sizeof(int));
		hipMalloc((void**)&values_dev, arraySize * sizeof(int));
		hipMalloc((void**)&w, 32768 * sizeof(int));
		hipMemcpy(wight_dev, wight, arraySize * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(values_dev, values, arraySize * sizeof(int), hipMemcpyHostToDevice);

		hipEvent_t gpu_start, gpu_stop;
		hipEventCreate(&gpu_start);
		hipEventCreate(&gpu_stop);
		hipEventRecord(gpu_start, 0);

		T_binary3 << <32768, 15 >> > (bin_dev, _del);

		hipEventRecord(gpu_stop, 0);
		hipEventSynchronize(gpu_stop);
		hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
		hipEventDestroy(gpu_start);
		hipEventDestroy(gpu_stop);
		cout << "T_binary3: " << gpu_elapsed_time << " milli-seconds\n";
sumoftime+=gpu_elapsed_time;
gpu_elapsed_time = 0;


		hipEventCreate(&gpu_start);
		hipEventCreate(&gpu_stop);
		hipEventRecord(gpu_start, 0);

		smth << <32768,15 >> > (bin_dev, wight_dev, s_dev, values_dev);

		hipEventRecord(gpu_stop, 0);
		hipEventSynchronize(gpu_stop);
		hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
		hipEventDestroy(gpu_start);
		hipEventDestroy(gpu_stop);
		cout << "smth: " << gpu_elapsed_time << " milli-seconds\n";
sumoftime+=gpu_elapsed_time;
gpu_elapsed_time = 0;


		hipEventCreate(&gpu_start);
		hipEventCreate(&gpu_stop);
		hipEventRecord(gpu_start, 0);

		plusing2_w << <32768, 15 >> > (bin_dev, w);

		hipEventRecord(gpu_stop, 0);
		hipEventSynchronize(gpu_stop);
		hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
		hipEventDestroy(gpu_start);
		hipEventDestroy(gpu_stop);
		cout << "plusing_weight: " << gpu_elapsed_time << " milli-seconds\n";
sumoftime+=gpu_elapsed_time;
gpu_elapsed_time = 0;


		hipEventCreate(&gpu_start);
		hipEventCreate(&gpu_stop);
		hipEventRecord(gpu_start, 0);

		plusing2_w << <32768, 15 >> > (s_dev, s);

		hipEventRecord(gpu_stop, 0);
		hipEventSynchronize(gpu_stop);
		hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
		hipEventDestroy(gpu_start);
		hipEventDestroy(gpu_stop);
		cout << "plusing_prices: " << gpu_elapsed_time << " milli-seconds\n";
sumoftime+=gpu_elapsed_time;
gpu_elapsed_time = 0;


		hipEventCreate(&gpu_start);
		hipEventCreate(&gpu_stop);
		hipEventRecord(gpu_start, 0);

		zeroing << <32, 1024 >> > (w, s, W);

		hipEventRecord(gpu_stop, 0);
		hipEventSynchronize(gpu_stop);
		hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
		hipEventDestroy(gpu_start);
		hipEventDestroy(gpu_stop);
		cout << "zeroing: " << gpu_elapsed_time << " milli-seconds\n";
sumoftime+=gpu_elapsed_time;
gpu_elapsed_time = 0;


		hipEventCreate(&gpu_start);
		hipEventCreate(&gpu_stop);
		hipEventRecord(gpu_start, 0);

		max1 << <32, 1024 >> > (s);

		hipEventRecord(gpu_stop, 0);
		hipEventSynchronize(gpu_stop);
		hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
		hipEventDestroy(gpu_start);
		hipEventDestroy(gpu_stop);
		cout << "max1: " << gpu_elapsed_time << " milli-seconds\n";
sumoftime+=gpu_elapsed_time;
gpu_elapsed_time = 0;



		hipEventCreate(&gpu_start);
		hipEventCreate(&gpu_stop);
		hipEventRecord(gpu_start, 0);

		for (int i = 32; i >= 1; i /= 2) {
			kermax2 << <1, i >> > (s,i);
			//hipDeviceSynchronize();

		}

		hipEventRecord(gpu_stop, 0);
		hipEventSynchronize(gpu_stop);
		hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
		hipEventDestroy(gpu_start);
		hipEventDestroy(gpu_stop);
		cout << "loops of kermax2: " << gpu_elapsed_time << " milli-seconds\n";
sumoftime+=gpu_elapsed_time;
gpu_elapsed_time = 0;


		hipEventCreate(&gpu_start);
		hipEventCreate(&gpu_stop);
		hipEventRecord(gpu_start, 0);

		hipMemcpy(Sum, s, sizeof(int), hipMemcpyDeviceToHost);

		hipEventRecord(gpu_stop, 0);
		hipEventSynchronize(gpu_stop);
		hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
		hipEventDestroy(gpu_start);
		hipEventDestroy(gpu_stop);
		cout << "cpy of maximal sum: " << gpu_elapsed_time << " milli-seconds\n";
sumoftime+=gpu_elapsed_time;
gpu_elapsed_time = 0;
		cout <<"\n"<<"max = " << Sum[0] << "\n";

		
		cout << "Total time: " << sumoftime << " milli-seconds\n";

		hipFree(bin_dev);
		hipFree(wight_dev);
		hipFree(s);
		hipFree(w);
		hipFree(s_dev);
		cout << "\n";


		//system("pause");

		//CPU version 1
		float fTimeStart = clock() / (float)(CLOCKS_PER_SEC);
		int bin[32768][15] = { 0 };
		int k = 0;
		for (int i = 0; i < 32768; i++) {
			k = i;
			for (int j = 14; j >= 0; j--) {
				bin[i][j] = k % 2;
				k /= 2;
			}
		}
		//int wight[15] = { 5,10,17,19,20, 23,26,30,32,38,   40,44,47,50,55 };// 55, 56, 60, 62, 66, 70	};
		//int values[15] = { 10,13,16,22,30, 25,55,90,110,115, 130,120,150,170,194 };// , 194, 199, 217, 230, 248	};
		int prices[32768][15] = { 0 };
		int weights[32768][15] = { 0 };
		int Sweig[32768] = { 0 };
		int Sval[32768] = { 0 };
		for (int i = 0; i < 32768; i++) {
			for (int j = 0; j < 15; j++) {
				weights[i][j] = wight[j] * bin[i][j];
				prices[i][j] = values[j] * bin[i][j];
			}
		}
		for (int i = 0; i < 32768; i++) {
			for (int j = 0; j < 15; j++) {
				Sweig[i] += weights[i][j];
				Sval[i] += prices[i][j];
			}
		}
		int max = 0; k = 0;
		for (int i = 0; i < 32768; i++) {
			if ((Sweig[i] <= W) && (Sval[i] > max)) {
				k = i; max = Sval[i];
			}
		}
		float fTimeStop = clock() / (float)CLOCKS_PER_SEC;
		cout << "\nmax = " << max << "\n" << "String No " << k << "\n";
		//for (int i = 0; i < 10; i++) { cout << bin[k][i] << " "; }cout << "\n";
		//for (int i = 0; i < 10; i++) { if (weights[k][i] != 0)cout << weights[k][i] << " + "; }cout << "\n";
		cout << "CPU time is " << (fTimeStop - fTimeStart) * 1000 << " milli-seconds\n";

		qwerty++;
	}
	

	return 0;
}

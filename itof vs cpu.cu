
#include "hip/hip_runtime.h"

#include "hip/device_functions.h"
#include <cstdlib>
#include <stdlib.h>
#include <sys/time.h>
#include <iostream>

#define arraySize 31 //35 max
#define threads_per_block 1024
#define max_blocks 32768
using namespace std;

__constant__ float coefs[arraySize*2];
__global__ void single_thread(float *sh_sum_dev,float W, float *str_num_dev, float num_of_blocks,int rep)
{
  register float th_w_sum = 0;
  register float th_v_sum = 0;
  register float th_bin[arraySize];
  register int n_of_it=rep;

  __shared__ float sh_maxs[threads_per_block];
  __shared__ float indices[threads_per_block];
  indices[threadIdx.x] = threadIdx.x;
  __syncthreads();

long int num_to_bin = blockIdx.x * blockDim.x + threadIdx.x;
num_to_bin += max_blocks * n_of_it;
#pragma unroll
  for (uint i = 0; i < arraySize; i++)
    {
      th_bin[i] = ((num_to_bin) >> i) % 2;

      th_w_sum += th_bin[i] * coefs[i];
      th_v_sum += th_bin[i] * coefs[i+arraySize];
    }

sh_maxs[threadIdx.x] = (th_w_sum > W) ? 0:th_v_sum;

__syncthreads ();
  for (uint offset = blockDim.x / 2; offset >= 1; offset >>= 1)
    {
      if (threadIdx.x < offset)
	{
	  if (sh_maxs[threadIdx.x] < sh_maxs[threadIdx.x + offset])
	    {
	      sh_maxs[threadIdx.x] = sh_maxs[threadIdx.x + offset];
	      indices[threadIdx.x] = indices[threadIdx.x + offset];
	    }
	}
      __syncthreads ();
    }
  // write result for this block to global mem
  if(threadIdx.x == 0){
  sh_sum_dev[blockIdx.x+max_blocks*rep] = sh_maxs[0];
  str_num_dev[blockIdx.x+max_blocks*rep] = indices[0] + blockIdx.x * blockDim.x +max_blocks*rep;
}}

__global__ void
reduction_max (float *s, float *str_num_dev)
{
  int ID = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ float sdata[threads_per_block*2];
  sdata[threadIdx.x] = s[ID];
  sdata[threadIdx.x + threads_per_block] = str_num_dev[ID];

  __syncthreads ();
  // do reduction in shared mem
  for (uint s = blockDim.x / 2; s > 0; s >>= 1)
    {
      if (threadIdx.x < s)
	{
	  if (sdata[threadIdx.x] < sdata[threadIdx.x + s])
	    {
	      sdata[threadIdx.x] = sdata[threadIdx.x + s];
	      //sdata[tid+64] = sdata[tid + s+64];
	      sdata[threadIdx.x + threads_per_block] =
		sdata[threadIdx.x + threads_per_block + s];
	    }
	}
      __syncthreads ();
    }
  // write result for this block to global mem
  if (threadIdx.x == 0)
    {
			//if(sdata[0]>s[0]){//}&&(blockIdx.x>0)){
      s[blockIdx.x] = sdata[0];
      str_num_dev[blockIdx.x] = sdata[threads_per_block];
		}
    //}
}

__global__ void
which_string (int a, int *view_dev)
{
  view_dev[threadIdx.x] = (a >> threadIdx.x) % 2;
}

int main(){
  float W = 500;

  struct timeval t0, t1;
  hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);
float milliseconds1 = 0;
float milliseconds2 = 0;
  long int strSize_b = pow (2, arraySize);
  int num_of_blocks = strSize_b / threads_per_block;
  float *Sum = new float[1];	// = { 0 };
  float *sh_sum_dev;
  float weight[31] ={ 5, 10, 17, 19, 20, 23, 26, 30, 32, 38, 40, 44, 47, 50, 55, 56, 56, 60, 62, 66, 70, 75, 77, 80, 81, 90,93,96,101,107,115 };
  float values[31] ={ 10, 13, 16, 22, 30, 25, 55, 90, 110, 115, 130, 120, 150, 170, 194, 199, 194, 199, 217, 230, 248, 250, 264, 271, 279, 286,293,299,305,313,321 };
  float dev_coefs[62] = {5, 10, 17, 19, 20, 23, 26, 30, 32, 38, 40, 44, 47, 50, 55, 56, 56, 60, 62, 66, 70, 75, 77, 80, 81, 90,93,96,101,107,115, 10, 13, 16, 22, 30, 25, 55, 90, 110, 115, 130, 120, 150, 170, 194, 199, 194, 199, 217, 230, 248, 250, 264, 271, 279, 286,293,299,305,313,321 };
  //float dev_coefs[60] = {5, 10, 17, 19, 20, 23, 26, 30, 32, 38, 40, 44, 47, 50, 55, 56, 56, 60, 62, 66, 70, 75, 77, 80, 81, 90,93,96,101,107, 10, 13, 16, 22, 30, 25, 55, 90, 110, 115, 130, 120, 150, 170, 194, 199, 194, 199, 217, 230, 248, 250, 264, 271, 279, 286,293,299,305,313 };

  //float *values_dev;
  long sec, usec;
  float *str_num_dev;
  float *str_num = new float[1];
  float N_of_rep;
  N_of_rep = num_of_blocks/max_blocks;

  cout <<"N of items "<<arraySize<<"\n";
  cout<<"N of blocks "<<num_of_blocks<<"\n";
  cout<<"strSize_b = "<<strSize_b<<"\n";
  cout<<"num_of_blocks / threads_per_block = "<<num_of_blocks / threads_per_block<<"\n";
  cout<<"N of repeats = "<<N_of_rep<<"\n";
  cout<<"sing param = "<<num_of_blocks/N_of_rep<<" _ "<< threads_per_block<<"\n";
  cout<<"red param "<<num_of_blocks / threads_per_block<<"  ,  "<<strSize_b/num_of_blocks<<"\n";


  gettimeofday (&t0, NULL);
  hipMalloc ((void **) &sh_sum_dev,  num_of_blocks * sizeof (float));


  hipMalloc ((void **) &str_num_dev, num_of_blocks * sizeof (float));

  hipMemcpyToSymbol (HIP_SYMBOL(coefs), dev_coefs, 2*arraySize * sizeof (float));

hipEventRecord(start);
int sing_blocks = num_of_blocks/N_of_rep;
        for(int i = 0;i<N_of_rep;i++){
  single_thread <<< sing_blocks, threads_per_block >>> (sh_sum_dev, W, str_num_dev, num_of_blocks,i);
             }
hipEventRecord(stop);
hipEventSynchronize(stop);

hipEventElapsedTime(&milliseconds1, start, stop);
hipEventRecord(start);
int k = num_of_blocks/threads_per_block;
while(k>1){

               reduction_max <<<k, threads_per_block>>> (sh_sum_dev, str_num_dev);
               if(k>=threads_per_block){k/=threads_per_block;}
               else break;
             }

reduction_max <<<1,k>>> (sh_sum_dev, str_num_dev);
hipEventSynchronize(stop);

hipEventElapsedTime(&milliseconds2, start, stop);
  hipMemcpy (Sum, sh_sum_dev, sizeof (float), hipMemcpyDeviceToHost);
  hipMemcpy (str_num, str_num_dev, sizeof (float), hipMemcpyDeviceToHost);
	gettimeofday (&t1, 0);
	sec = (t1.tv_sec - t0.tv_sec);
  usec = t1.tv_usec - t0.tv_usec;
  cout << "GPU time = " << sec << " sec, " << usec << " microsec\n";
  cout<<"single_thread time = "<<milliseconds1/1000.0<<"sec\n";
  cout<<"reduction time = "<<milliseconds2/1000.0<<"sec\n";

  cout << "Acheived maximal sum = " << Sum[0] << "\n";
  cout << "String number " << str_num[0] << "\n";

  int *view = new int[arraySize];
  int *view_dev;
  hipMalloc ((void **) &view_dev, arraySize * sizeof (int));
  which_string <<< 1, arraySize >>> (str_num[0], view_dev);
  hipMemcpy (view, view_dev, arraySize * sizeof (int),
	      hipMemcpyDeviceToHost);
  for (int i = 0; i < arraySize; i++)
    {
      cout << view[i] << " ";
    } cout << "\n";
  //check
  float checksum = 0;
  for (int i = 0; i < arraySize; i++)
    {
      checksum += values[i] * view[i];
    } cout << "Validation sum = " << checksum << "\n";
  checksum = 0;
  for (int i = 0; i < arraySize; i++)
    {
      checksum += weight[i] * view[i];
    } cout << "Weight = " << checksum << "\n";
  hipFree (sh_sum_dev);
  hipFree (str_num_dev);
  hipFree (coefs);
  hipFree (view_dev);


cout<<"CPU version:\n";

float *cpu_bin = new float[arraySize];
int max = 0;
int tmp = 0;
int cpu_str = 0;
int cap;
gettimeofday (&t0, NULL);
for(long int i = 0;i<num_of_blocks;i++){
  for(int j = 0; j<threads_per_block;j++){
    int tobin = i*threads_per_block+j;
    for(int k = 0; k<arraySize;k++){
      cpu_bin[k] = tobin%2;
      tobin>>=1;
      tmp += cpu_bin[k]*values[k];
      cap += cpu_bin[k]*weight[k];
    }
    if((cap<=W)&&(tmp>max)){max = tmp;cpu_str = i*threads_per_block+j;}
    tmp = 0; cap = 0;
}
}

gettimeofday (&t1, 0);
sec = (t1.tv_sec - t0.tv_sec);
usec = t1.tv_usec - t0.tv_usec;
cout<<"Max = "<<max<<"\n"<<"STR = "<<cpu_str<<"\n";
cout << "CPU time = " << sec << " sec, " << usec << " microsec\n";

  return 0;
}
